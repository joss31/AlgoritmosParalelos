#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <fcntl.h>
#include <stdlib.h>
#include <iostream>
#include "lodepng.h"
using namespace std;

__global__
void PictureKernell(unsigned char* d_Pin, unsigned char* d_Pout, int n, int m){
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int new_pos;
  if((y < n) && (x < m)) {
    new_pos = (y*m+x)*4;
    d_Pout[new_pos] = 2*d_Pin[new_pos];
    d_Pout[new_pos+1] = 2*d_Pin[new_pos+1];
    d_Pout[new_pos+2] = 2*d_Pin[new_pos+2];
    d_Pout[new_pos+3] = d_Pin[new_pos+3];
  }
}

__global__
void PictureKernelll(unsigned char* d_Pin, unsigned char* d_Pout, int n, int m){
	/*int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	if ((Row < height) && (Col < width)) {
		d_Pout[Row*width+Col] = 2.0*d_Pin[Row*width+Col];
	}
  */
  
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int new_pos;
  if((y < n) && (x < m)) {
    new_pos = (y*m+x);
    d_Pout[new_pos] = 2*d_Pin[new_pos];
  }
}


__global__
void PictureKernel1D(unsigned char* d_Pin, unsigned char* d_Pout, int n, int m){
  int Row = blockIdx.x * blockDim.x + threadIdx.x;
  if(Row < n*m*4) {
    d_Pout[Row] = 2*d_Pin[Row];
  }
}

void Picture(unsigned char* Pin, unsigned char* Pout, int n, int m){
  unsigned char* d_Pout, *d_Pin;
  long int size = n*m*4;
  hipMalloc((void **) &d_Pin,size);
  hipMemcpy(d_Pin, Pin, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &d_Pout,size);

   dim3 gridDim((m-1)/8+1,(n-1)/16+1,1);
   dim3 blockDim(8,16,1);
   //dim3 gridDim2((m-1)/8+1,(n-1)/16+1,1);
   //dim3 blockDim2(8,16,1);
   PictureKernell<<<gridDim,blockDim>>>(d_Pin,d_Pout,n,m);
   //PictureKernelll<<< gridDim2 , blockDim2 >>>(d_Pin,d_Pout,n,m);
  //PictureKernel1D<<<(size-1)/256+1,256>>>(d_Pin,d_Pout,n,m);

  hipMemcpy(Pout, d_Pout, size, hipMemcpyDeviceToHost);
  hipFree(d_Pin); hipFree(d_Pout);
}

int main(int argc, char * argv[] ){
  unsigned char *image, *out_image;
  int i;
  char name_in[100], name_out[100];
  unsigned width, height;
  if(argv[1] == NULL or argv[2] == NULL)
    cout << "Usage\n inverse.cu [input image] [output image]\n";
  strcpy(name_in,argv[1]);
  strcpy(name_out,argv[2]);
  i = lodepng_decode32_file(&image, &width, &height, name_in);
  if(i < 0) printf("NO\n");
  out_image = (unsigned char*) malloc(width*height*4);
  Picture(image,out_image,height,width);
  lodepng_encode32_file(name_out,out_image,width,height);

  free(image);
  free(out_image);
  return 0;
}
